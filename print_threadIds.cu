#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_threadIDs() {
  printf("threadIdx.x=%d, threadIdx.y=%d, threadIdx.z=%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
  int nx = 16;
  int ny = 4;
  //2-D grid (2x2), each block is also 2-D 8x2
  dim3 block(8, 2, 1);
  dim3 grid(nx / block.x, ny / block.y);

  print_threadIDs << <grid, block>> > ();

  //Wait until kernel call is completed
  hipDeviceSynchronize();

  //Reset the device
  hipDeviceReset();

  return 0;
}