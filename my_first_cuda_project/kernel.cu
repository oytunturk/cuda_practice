#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    hipError_t cudaStatus = addWithCuda(c , a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", 
        c[0], c[1], c[2], c[3], c[4]);
    
    //hipDeviceReset must be called before exiting in order for porifling and tracing 
    // tools such as Nsight and Visual Profiles to show complete traces 
    cudaStatus = cudeDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {

}