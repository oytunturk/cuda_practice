#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_world() {
  printf("Hello CUDA\n");
}

int main() {
  hello_world << <1, 1>> > ();

  //Wait until kernel call is completed
  hipDeviceSynchronize();
  
  //Reset the device
  hipDeviceReset();

  return 0;
}