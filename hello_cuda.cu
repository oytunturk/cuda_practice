#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_world() {
  printf("Hello CUDA\n");
}

int main() {
  //Runs it once (grid x block)
  //hello_world << <1, 1>> > ();
  //Runs it 20 times
  hello_world << <1, 20>> > ();

  //Wait until kernel call is completed
  hipDeviceSynchronize();
  
  //Reset the device
  hipDeviceReset();

  return 0;
}