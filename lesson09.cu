#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//Access data with thread indices --> this is very common in CUDA
__global__ void unique_idx_calc_threadIdx(int *input) {
  int tid = threadIdx.x;
  printf("input[%d]=%d\n", tid, input[tid]);
}

int main() {
  int arr[] = {10, 20, 30, 40, 50, 60, 70, 80};
  int *arr_d = NULL;
  int arr_size = 8;
  int nx = arr_size;
  int ny = 1;
  int nz = 1;
  //2-D grid (4x4x4), each block is 2x2x2
  dim3 block(8, 1, 1);
  dim3 grid(nx / block.x, ny / block.y, nz/block.z);

  size_t arr_byte_size = arr_size * sizeof(int);
  hipMalloc((void**)(&arr_d), arr_byte_size);
  hipMemcpy(arr_d, arr, arr_byte_size, hipMemcpyHostToDevice);

  unique_idx_calc_threadIdx << <grid, block>> > (arr_d);

  //Wait until kernel call is completed
  hipDeviceSynchronize();

  hipFree((void*)arr_d);

  //Reset the device
  hipDeviceReset();

  return 0;
}