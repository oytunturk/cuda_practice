#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_world() {
  printf("Hello CUDA\n");
}

int main() {
  int nx = 16;
  int ny = 4;
  //2-D grid (2x2), each block is also 2-D 8x2
  dim3 block(8, 2, 1);
  dim3 grid(nx / block.x, ny / block.y);
  
  hello_world << <grid, block>> > ();

  //Wait until kernel call is completed
  hipDeviceSynchronize();
  
  //Reset the device
  hipDeviceReset();

  return 0;
}