#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_world() {
  printf("Hello CUDA\n");
}

int main() {
  //Runs it once (grid dimension x block dimension)
  //Note that both grid and block can be up to 3-dimensions
  //hello_world << <1, 1>> > ();
  //Runs it 20 times
  hello_world << <1, 20>> > ();

  //Wait until kernel call is completed
  hipDeviceSynchronize();
  
  //Reset the device
  hipDeviceReset();

  return 0;
}